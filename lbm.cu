#include "hip/hip_runtime.h"
// the following additional constants need to be defined:
// LAT_H, LAT_W, BLOCK_SIZE, GEO_FLUID, GEO_WALL, GEO_INFLOW

#define DT 1.0f

__constant__ float tau;			// relaxation time

struct DistP {
	float *fC, *fE, *fW, *fS, *fN, *fSE, *fSW, *fNE, *fNW;
};

struct Dist {
	float fC, fE, fW, fS, fN, fSE, fSW, fNE, fNW;
};

__device__ void inline getDist(Dist &dout, DistP din, int idx)
{
	dout.fC = din.fC[idx];
	dout.fE = din.fE[idx];
	dout.fW = din.fW[idx];
	dout.fS = din.fS[idx];
	dout.fN = din.fN[idx];
	dout.fNE = din.fNE[idx];
	dout.fNW = din.fNW[idx];
	dout.fSE = din.fSE[idx];
	dout.fSW = din.fSW[idx];
}

__device__ void inline getMacro(Dist fi, int *map, int idx, float &rho, float2 &v)
{
	rho = fi.fC + fi.fE + fi.fW + fi.fS + fi.fN + fi.fNE + fi.fNW + fi.fSE + fi.fSW;
	if (map[idx] == GEO_INFLOW) {
		v.x = 0.1f;
		v.y = 0.0f;
	} else {
		v.x = (fi.fE + fi.fSE + fi.fNE - fi.fW - fi.fSW - fi.fNW) / rho;
		v.y = (fi.fN + fi.fNW + fi.fNE - fi.fS - fi.fSW - fi.fSE) / rho;
	}
}

__global__ void LBMUpdateTracerParticles(DistP cd, int *map, float *x, float *y)
{
	float rho;
	float2 pv;

	int gi = threadIdx.x + blockDim.x * blockIdx.x;
	float cx = x[gi];
	float cy = y[gi];

	int ix = (int)(cx);
	int iy = (int)(cy);

	if (iy < 0)
		iy = 0;

	if (ix < 0)
		ix = 0;

	if (ix > LAT_W-1)
		ix = LAT_W-1;

	if (iy > LAT_H-1)
		iy = LAT_H-1;

	int dix = ix + LAT_W*iy;

	Dist fc;
	getDist(fc, cd, dix);
	getMacro(fc, map, dix, rho, pv);

	cx = cx + pv.x * DT;
	cy = cy + pv.y * DT;

	if (cx > LAT_W)
		cx = 0.0f;

	if (cy > LAT_H)
		cy = 0.0f;

	if (cx < 0.0f)
		cx = (float)LAT_W;

	if (cy < 0.0f)
		cy = (float)LAT_H;

	x[gi] = cx;
	y[gi] = cy;
}

// TODO:
// - try having dummy nodes as the edges of the lattice to avoid divergent threads

__global__ void LBMCollideAndPropagate(int *map, DistP cd, DistP od, float *orho, float *ovx, float *ovy)
{
	int tix = threadIdx.x;
	int ti = tix + blockIdx.x * blockDim.x;
	int gi = ti + LAT_W*blockIdx.y;

	// equilibrium distributions
	Dist feq, fi;
	float rho;
	float2 v;

	// shared variables for in-block propagation
	__shared__ float fo_E[BLOCK_SIZE];
	__shared__ float fo_W[BLOCK_SIZE];
	__shared__ float fo_SE[BLOCK_SIZE];
	__shared__ float fo_SW[BLOCK_SIZE];
	__shared__ float fo_NE[BLOCK_SIZE];
	__shared__ float fo_NW[BLOCK_SIZE];

	// cache the distribution in local variables
	getDist(fi, cd, gi);

	// macroscopic quantities for the current cell
	getMacro(fi, map, gi, rho, v);

	if (orho != NULL) {
		orho[gi] = rho;
		ovx[gi] = v.x;
		ovy[gi] = v.y;
	}

	// relaxation
	float Cusq = -1.5f * (v.x*v.x + v.y*v.y);

	feq.fC = rho * (1.0f + Cusq) * 4.0f/9.0f;
	feq.fN = rho * (1.0f + Cusq + 3.0f*v.y + 4.5f*v.y*v.y) / 9.0f;
	feq.fE = rho * (1.0f + Cusq + 3.0f*v.x + 4.5f*v.x*v.x) / 9.0f;
	feq.fS = rho * (1.0f + Cusq - 3.0f*v.y + 4.5f*v.y*v.y) / 9.0f;
	feq.fW = rho * (1.0f + Cusq - 3.0f*v.x + 4.5f*v.x*v.x) / 9.0f;
	feq.fNE = rho * (1.0f + Cusq + 3.0f*(v.x+v.y) + 4.5f*(v.x+v.y)*(v.x+v.y)) / 36.0f;
	feq.fSE = rho * (1.0f + Cusq + 3.0f*(v.x-v.y) + 4.5f*(v.x-v.y)*(v.x-v.y)) / 36.0f;
	feq.fSW = rho * (1.0f + Cusq + 3.0f*(-v.x-v.y) + 4.5f*(v.x+v.y)*(v.x+v.y)) / 36.0f;
	feq.fNW = rho * (1.0f + Cusq + 3.0f*(-v.x+v.y) + 4.5f*(-v.x+v.y)*(-v.x+v.y)) / 36.0f;

	if (map[gi] == GEO_FLUID) {
		fi.fC += (feq.fC - fi.fC) / tau;
		fi.fE += (feq.fE - fi.fE) / tau;
		fi.fW += (feq.fW - fi.fW) / tau;
		fi.fS += (feq.fS - fi.fS) / tau;
		fi.fN += (feq.fN - fi.fN) / tau;
		fi.fSE += (feq.fSE - fi.fSE) / tau;
		fi.fNE += (feq.fNE - fi.fNE) / tau;
		fi.fSW += (feq.fSW - fi.fSW) / tau;
		fi.fNW += (feq.fNW - fi.fNW) / tau;
	} else if (map[gi] == GEO_INFLOW) {
		fi.fC  = feq.fC;
		fi.fE  = feq.fE;
		fi.fW  = feq.fW;
		fi.fS  = feq.fS;
		fi.fN  = feq.fN;
		fi.fSE = feq.fSE;
		fi.fNE = feq.fNE;
		fi.fSW = feq.fSW;
		fi.fNW = feq.fNW;
	} else if (map[gi] == GEO_WALL) {
		float t;
		t = fi.fE;
		fi.fE = fi.fW;
		fi.fW = t;

		t = fi.fNW;
		fi.fNW = fi.fSE;
		fi.fSE = t;

		t = fi.fNE;
		fi.fNE = fi.fSW;
		fi.fSW = t;

		t = fi.fN;
		fi.fN = fi.fS;
		fi.fS = t;
	}

	od.fC[gi] = fi.fC;

	// N + S propagation (global memory)
	if (blockIdx.y > 0)			od.fS[gi-LAT_W] = fi.fS;
	if (blockIdx.y < LAT_H-1)	od.fN[gi+LAT_W] = fi.fN;

	// E propagation in shared memory
	if (tix < blockDim.x-1) {
		fo_E[tix+1] = fi.fE;
		fo_NE[tix+1] = fi.fNE;
		fo_SE[tix+1] = fi.fSE;
	// E propagation in global memory (at block boundary)
	} else if (ti < LAT_W) {
		od.fE[gi+1] = fi.fE;
		if (blockIdx.y > 0)			od.fSE[gi-LAT_W+1] = fi.fSE;
		if (blockIdx.y < LAT_H-1)	od.fNE[gi+LAT_W+1] = fi.fNE;
	}

	// W propagation in shared memory
	if (tix > 0) {
		fo_W[tix-1] = fi.fW;
		fo_NW[tix-1] = fi.fNW;
		fo_SW[tix-1] = fi.fSW;
	// W propagation in global memory (at block boundary)
	} else if (ti > 0) {
		od.fW[gi-1] = fi.fW;
		if (blockIdx.y > 0)			od.fSW[gi-LAT_W-1] = fi.fSW;
		if (blockIdx.y < LAT_H-1)	od.fNW[gi+LAT_W-1] = fi.fNW;
	}

	__syncthreads();

	// the leftmost thread is not updated in this block
	if (tix > 0) {
		od.fE[gi] = fo_E[tix];
		if (blockIdx.y > 0)			od.fSE[gi-LAT_W] = fo_SE[tix];
		if (blockIdx.y < LAT_H-1)	od.fNE[gi+LAT_W] = fo_NE[tix];
	}

	// the rightmost thread is not updated in this block
	if (tix < blockDim.x-1) {
		od.fW[gi] = fo_W[tix];
		if (blockIdx.y > 0)			od.fSW[gi-LAT_W] = fo_SW[tix];
		if (blockIdx.y < LAT_H-1)	od.fNW[gi+LAT_W] = fo_NW[tix];
	}
}

