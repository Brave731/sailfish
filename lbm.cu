#include "hip/hip_runtime.h"
// The following additional constants need to be defined:
// LAT_H, LAT_W, BLOCK_SIZE, GEO_FLUID, GEO_WALL, GEO_INFLOW

// If INFLOW_PROP is set, new distributions will be propagated into
// GEO_INFLOW nodes and thus the velocity of these nodes will have to
// be explicitly overridden at relaxation time.

#define RELAX_bgk	BGK_relaxate(rho, v, fi, map[gi]);
#define RELAX_mrt	MS_relaxate(fi, map[gi]);

#define DT 1.0f

__constant__ float tau;			// relaxation time
__constant__ float visc;		// viscosity

struct DistP {
	float *fC, *fE, *fW, *fS, *fN, *fSE, *fSW, *fNE, *fNW;
};

struct Dist {
	float fC, fE, fW, fS, fN, fSE, fSW, fNE, fNW;
};

// Distribution in momentum space.
struct DistM {
	float rho, en, ens, mx, ex, my, ey, sd, sod;
};

//
// Copy the idx-th distribution from din into dout.
//
__device__ void inline getDist(Dist &dout, DistP din, int idx)
{
	dout.fC = din.fC[idx];
	dout.fE = din.fE[idx];
	dout.fW = din.fW[idx];
	dout.fS = din.fS[idx];
	dout.fN = din.fN[idx];
	dout.fNE = din.fNE[idx];
	dout.fNW = din.fNW[idx];
	dout.fSE = din.fSE[idx];
	dout.fSW = din.fSW[idx];
}

//
// Get macroscopic density rho and velocity v given a distribution fi, and
// the node class node_type.
//
__device__ void inline getMacro(Dist fi, int node_type, float &rho, float2 &v)
{
	rho = fi.fC + fi.fE + fi.fW + fi.fS + fi.fN + fi.fNE + fi.fNW + fi.fSE + fi.fSW;
#ifdef INFLOW_PROP
	if (node_type == GEO_INFLOW) {
		v.x = 0.1f;
		v.y = 0.0f;
	} else
#endif
	{
		v.x = (fi.fE + fi.fSE + fi.fNE - fi.fW - fi.fSW - fi.fNW) / rho;
		v.y = (fi.fN + fi.fNW + fi.fNE - fi.fS - fi.fSW - fi.fSE) / rho;
	}
}

//
// A kernel to update the position of tracer particles.
//
// Each thread updates the position of a single particle using Euler's algorithm.
//
__global__ void LBMUpdateTracerParticles(DistP cd, int *map, float *x, float *y)
{
	float rho;
	float2 pv;

	int gi = threadIdx.x + blockDim.x * blockIdx.x;
	float cx = x[gi];
	float cy = y[gi];

	int ix = (int)(cx);
	int iy = (int)(cy);

	// Sanity checks.
	if (iy < 0)
		iy = 0;

	if (ix < 0)
		ix = 0;

	if (ix > LAT_W-1)
		ix = LAT_W-1;

	if (iy > LAT_H-1)
		iy = LAT_H-1;

	int dix = ix + LAT_W*iy;

	Dist fc;
	getDist(fc, cd, dix);
	getMacro(fc, map[dix], rho, pv);

	cx = cx + pv.x * DT;
	cy = cy + pv.y * DT;

	// Periodic boundary conditions.
	if (cx > LAT_W)
		cx = 0.0f;

	if (cy > LAT_H)
		cy = 0.0f;

	if (cx < 0.0f)
		cx = (float)LAT_W;

	if (cy < 0.0f)
		cy = (float)LAT_H;

	x[gi] = cx;
	y[gi] = cy;
}

//
// Relaxation in moment space.
//
__device__ void inline MS_relaxate(Dist &fi, int node_type)
{
	DistM fm, feq;

	fm.rho = 1.0f*fi.fC + 1.0f*fi.fE + 1.0f*fi.fN + 1.0f*fi.fW + 1.0f*fi.fS + 1.0f*fi.fNE + 1.0f*fi.fNW + 1.0f*fi.fSW + 1.0f*fi.fSE;
	fm.en = -4.0f*fi.fC - 1.0f*fi.fE - 1.0f*fi.fN - 1.0f*fi.fW - 1.0f*fi.fS + 2.0f*fi.fNE + 2.0f*fi.fNW + 2.0f*fi.fSW + 2.0f*fi.fSE;
	fm.ens = 4.0f*fi.fC - 2.0f*fi.fE - 2.0f*fi.fN - 2.0f*fi.fW - 2.0f*fi.fS + 1.0f*fi.fNE + 1.0f*fi.fNW + 1.0f*fi.fSW + 1.0f*fi.fSE;
	fm.mx =  0.0f*fi.fC + 1.0f*fi.fE + 0.0f*fi.fN - 1.0f*fi.fW + 0.0f*fi.fS + 1.0f*fi.fNE - 1.0f*fi.fNW - 1.0f*fi.fSW + 1.0f*fi.fSE;
	fm.ex =  0.0f*fi.fC - 2.0f*fi.fE + 0.0f*fi.fN + 2.0f*fi.fW + 0.0f*fi.fS + 1.0f*fi.fNE - 1.0f*fi.fNW - 1.0f*fi.fSW + 1.0f*fi.fSE;
	fm.my =  0.0f*fi.fC + 0.0f*fi.fE + 1.0f*fi.fN + 0.0f*fi.fW - 1.0f*fi.fS + 1.0f*fi.fNE + 1.0f*fi.fNW - 1.0f*fi.fSW - 1.0f*fi.fSE;
	fm.ey =  0.0f*fi.fC + 0.0f*fi.fE - 2.0f*fi.fN + 0.0f*fi.fW + 2.0f*fi.fS + 1.0f*fi.fNE + 1.0f*fi.fNW - 1.0f*fi.fSW - 1.0f*fi.fSE;
	fm.sd =  0.0f*fi.fC + 1.0f*fi.fE - 1.0f*fi.fN + 1.0f*fi.fW - 1.0f*fi.fS + 0.0f*fi.fNE + 0.0f*fi.fNW + 0.0f*fi.fSW - 0.0f*fi.fSE;
	fm.sod = 0.0f*fi.fC + 0.0f*fi.fE + 0.0f*fi.fN + 0.0f*fi.fW + 0.0f*fi.fS + 1.0f*fi.fNE - 1.0f*fi.fNW + 1.0f*fi.fSW - 1.0f*fi.fSE;

#ifdef INFLOW_PROP
	if (node_type == GEO_INFLOW) {
		fm.mx = 0.1f;
		fm.my = 0.0f;
	}
#endif

	float h = fm.mx*fm.mx + fm.my*fm.my;
	feq.en  = -2.0f*fm.rho + 3.0f*h;
	feq.ens = fm.rho - 3.0f*h;
	feq.ex  = -fm.mx;
	feq.ey  = -fm.my;
	feq.sd  = (fm.mx*fm.mx - fm.my*fm.my);
	feq.sod = (fm.mx*fm.my);

	float tau7 = 4.0f / (12.0f*visc + 2.0f);
	float tau4 = 3.0f*(2.0f - tau7) / (3.0f - tau7);
	float tau8 = 1.0f/((2.0f/tau7 - 1.0f)*0.5f + 0.5f);

	if (node_type == GEO_FLUID) {
		fm.en  -= 1.63f * (fm.en - feq.en);
		fm.ens -= 1.14f * (fm.ens - feq.ens);
		fm.ex  -= tau4 * (fm.ex - feq.ex);
		fm.ey  -= 1.92f * (fm.ey - feq.ey);
		fm.sd  -= tau7 * (fm.sd - feq.sd);
		fm.sod -= tau8 * (fm.sod - feq.sod);
	} else if (node_type == GEO_INFLOW) {
		fm.en  = feq.en;
		fm.ens = feq.ens;
		fm.ex  = feq.ex;
		fm.ey  = feq.ey;
		fm.sd  = feq.sd;
		fm.sod = feq.sod;
	} else if (node_type == GEO_WALL) {
		float t;
		t = fi.fE;
		fi.fE = fi.fW;
		fi.fW = t;

		t = fi.fNW;
		fi.fNW = fi.fSE;
		fi.fSE = t;

		t = fi.fNE;
		fi.fNE = fi.fSW;
		fi.fSW = t;

		t = fi.fN;
		fi.fN = fi.fS;
		fi.fS = t;
	}

	if (node_type != GEO_WALL) {
		fi.fC  = (1.0f/9.0f)*fm.rho - (1.0f/9.0f)*fm.en + (1.0f/9.0f)*fm.ens;
		fi.fE  = (1.0f/9.0f)*fm.rho - (1.0f/36.0f)*fm.en - (1.0f/18.0f)*fm.ens + (1.0f/6.0f)*fm.mx - (1.0f/6.0f)*fm.ex + 0.25f*fm.sd;
		fi.fN  = (1.0f/9.0f)*fm.rho - (1.0f/36.0f)*fm.en - (1.0f/18.0f)*fm.ens + (1.0f/6.0f)*fm.my - (1.0f/6.0f)*fm.ey - 0.25f*fm.sd;
		fi.fW  = (1.0f/9.0f)*fm.rho - (1.0f/36.0f)*fm.en - (1.0f/18.0f)*fm.ens - (1.0f/6.0f)*fm.mx + (1.0f/6.0f)*fm.ex + 0.25f*fm.sd;
		fi.fS  = (1.0f/9.0f)*fm.rho - (1.0f/36.0f)*fm.en - (1.0f/18.0f)*fm.ens - (1.0f/6.0f)*fm.my + (1.0f/6.0f)*fm.ey - 0.25f*fm.sd;
		fi.fNE = (1.0f/9.0f)*fm.rho + (1.0f/18.0f)*fm.en + (1.0f/36.0f)*fm.ens +
				 +(1.0f/6.0f)*fm.mx + (1.0f/12.0f)*fm.ex + (1.0f/6.0f)*fm.my + (1.0f/12.0f)*fm.ey + 0.25f*fm.sod;
		fi.fNW = (1.0f/9.0f)*fm.rho + (1.0f/18.0f)*fm.en + (1.0f/36.0f)*fm.ens +
				 -(1.0f/6.0f)*fm.mx - (1.0f/12.0f)*fm.ex + (1.0f/6.0f)*fm.my + (1.0f/12.0f)*fm.ey - 0.25f*fm.sod;
		fi.fSW = (1.0f/9.0f)*fm.rho + (1.0f/18.0f)*fm.en + (1.0f/36.0f)*fm.ens +
				 -(1.0f/6.0f)*fm.mx - (1.0f/12.0f)*fm.ex - (1.0f/6.0f)*fm.my - (1.0f/12.0f)*fm.ey + 0.25f*fm.sod;
		fi.fSE = (1.0f/9.0f)*fm.rho + (1.0f/18.0f)*fm.en + (1.0f/36.0f)*fm.ens +
				 +(1.0f/6.0f)*fm.mx + (1.0f/12.0f)*fm.ex - (1.0f/6.0f)*fm.my - (1.0f/12.0f)*fm.ey - 0.25f*fm.sod;
	}
}

//
// Performs the relaxation step in the BGK model given the density rho,
// the velocity v and the distribution fi.
//
__device__ void inline BGK_relaxate(float rho, float2 v, Dist &fi, int node_type)
{
	// relaxation
	float Cusq = -1.5f * (v.x*v.x + v.y*v.y);
	Dist feq;

	feq.fC = rho * (1.0f + Cusq) * 4.0f/9.0f;
	feq.fN = rho * (1.0f + Cusq + 3.0f*v.y + 4.5f*v.y*v.y) / 9.0f;
	feq.fE = rho * (1.0f + Cusq + 3.0f*v.x + 4.5f*v.x*v.x) / 9.0f;
	feq.fS = rho * (1.0f + Cusq - 3.0f*v.y + 4.5f*v.y*v.y) / 9.0f;
	feq.fW = rho * (1.0f + Cusq - 3.0f*v.x + 4.5f*v.x*v.x) / 9.0f;
	feq.fNE = rho * (1.0f + Cusq + 3.0f*(v.x+v.y) + 4.5f*(v.x+v.y)*(v.x+v.y)) / 36.0f;
	feq.fSE = rho * (1.0f + Cusq + 3.0f*(v.x-v.y) + 4.5f*(v.x-v.y)*(v.x-v.y)) / 36.0f;
	feq.fSW = rho * (1.0f + Cusq + 3.0f*(-v.x-v.y) + 4.5f*(v.x+v.y)*(v.x+v.y)) / 36.0f;
	feq.fNW = rho * (1.0f + Cusq + 3.0f*(-v.x+v.y) + 4.5f*(-v.x+v.y)*(-v.x+v.y)) / 36.0f;

	if (node_type == GEO_FLUID) {
		fi.fC += (feq.fC - fi.fC) / tau;
		fi.fE += (feq.fE - fi.fE) / tau;
		fi.fW += (feq.fW - fi.fW) / tau;
		fi.fS += (feq.fS - fi.fS) / tau;
		fi.fN += (feq.fN - fi.fN) / tau;
		fi.fSE += (feq.fSE - fi.fSE) / tau;
		fi.fNE += (feq.fNE - fi.fNE) / tau;
		fi.fSW += (feq.fSW - fi.fSW) / tau;
		fi.fNW += (feq.fNW - fi.fNW) / tau;
	} else if (node_type == GEO_INFLOW) {
		fi.fC  = feq.fC;
		fi.fE  = feq.fE;
		fi.fW  = feq.fW;
		fi.fS  = feq.fS;
		fi.fN  = feq.fN;
		fi.fSE = feq.fSE;
		fi.fNE = feq.fNE;
		fi.fSW = feq.fSW;
		fi.fNW = feq.fNW;
	} else if (node_type == GEO_WALL) {
		float t;
		t = fi.fE;
		fi.fE = fi.fW;
		fi.fW = t;

		t = fi.fNW;
		fi.fNW = fi.fSE;
		fi.fSE = t;

		t = fi.fNE;
		fi.fNE = fi.fSW;
		fi.fSW = t;

		t = fi.fN;
		fi.fN = fi.fS;
		fi.fS = t;
	}
}

// TODO:
// - try having dummy nodes as the edges of the lattice to avoid divergent threads

__global__ void LBMCollideAndPropagate(int *map, DistP cd, DistP od, float *orho, float *ovx, float *ovy)
{
	int tix = threadIdx.x;
	int ti = tix + blockIdx.x * blockDim.x;
	int gi = ti + LAT_W*blockIdx.y;

	// shared variables for in-block propagation
	__shared__ float fo_E[BLOCK_SIZE];
	__shared__ float fo_W[BLOCK_SIZE];
	__shared__ float fo_SE[BLOCK_SIZE];
	__shared__ float fo_SW[BLOCK_SIZE];
	__shared__ float fo_NE[BLOCK_SIZE];
	__shared__ float fo_NW[BLOCK_SIZE];

	// cache the distribution in local variables
	Dist fi;
	getDist(fi, cd, gi);

	// macroscopic quantities for the current cell
	float rho;
	float2 v;
	getMacro(fi, map[gi], rho, v);

	// only save the macroscopic quantities if requested to do so
	if (orho != NULL) {
		orho[gi] = rho;
		ovx[gi] = v.x;
		ovy[gi] = v.y;
	}

	RELAXATE;

	// update the 0-th direction distribution
	od.fC[gi] = fi.fC;

#ifdef INFLOW_PROP
	#define set_odist(idx, dir, val, mtype) od.dir[idx] = val;
#else
	#define set_odist(idx, dir, val, mtype) if (mtype != GEO_INFLOW) { od.dir[idx] = val; }
#endif

	// E propagation in shared memory
	if (tix < blockDim.x-1) {
		fo_E[tix+1] = fi.fE;
		fo_NE[tix+1] = fi.fNE;
		fo_SE[tix+1] = fi.fSE;
	// E propagation in global memory (at right block boundary)
	} else if (ti < LAT_W) {
		set_odist(gi+1, fE, fi.fE, map[gi+1]);
		if (blockIdx.y > 0)			set_odist(gi-LAT_W+1, fSE, fi.fSE, map[gi-LAT_W+1]);
		if (blockIdx.y < LAT_H-1)	set_odist(gi+LAT_W+1, fNE, fi.fNE, map[gi+LAT_W+1]);
	}

	// W propagation in shared memory
	if (tix > 0) {
		fo_W[tix-1] = fi.fW;
		fo_NW[tix-1] = fi.fNW;
		fo_SW[tix-1] = fi.fSW;
	// W propagation in global memory (at left block boundary)
	} else if (ti > 0) {
		set_odist(gi-1, fW, fi.fW, map[gi-1]);
		if (blockIdx.y > 0)			set_odist(gi-LAT_W-1, fSW, fi.fSW, map[gi-LAT_W-1]);
		if (blockIdx.y < LAT_H-1)	set_odist(gi+LAT_W-1, fNW, fi.fNW, map[gi+LAT_W-1]);
	}

	__syncthreads();

#ifndef INFLOW_PROP
	int m1 = map[gi];
	int m2 = map[gi-LAT_W];
	int m3 = map[gi+LAT_W];
#endif

	// the leftmost thread is not updated in this block
	if (tix > 0) {
		set_odist(gi, fE, fo_E[tix], m1);
		if (blockIdx.y > 0)			set_odist(gi-LAT_W, fSE, fo_SE[tix], m2);
		if (blockIdx.y < LAT_H-1)	set_odist(gi+LAT_W, fNE, fo_NE[tix], m3);
	}

	// N + S propagation (global memory)
	if (blockIdx.y > 0)			set_odist(gi-LAT_W, fS, fi.fS, m2);
	if (blockIdx.y < LAT_H-1)	set_odist(gi+LAT_W, fN, fi.fN, m3);

	// the rightmost thread is not updated in this block
	if (tix < blockDim.x-1) {
		set_odist(gi, fW, fo_W[tix], m1);
		if (blockIdx.y > 0)			set_odist(gi-LAT_W, fSW, fo_SW[tix], m2);
		if (blockIdx.y < LAT_H-1)	set_odist(gi+LAT_W, fNW, fo_NW[tix], m3);
	}
}

